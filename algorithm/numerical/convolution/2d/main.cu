#include <iostream>
#include <random>
#include <iomanip>
#include <hip/hip_runtime.h>

#define MASK_DIM_SIZE 10 // Number of elements for one spacial dimension
#define TILE_SIZE 16

__constant__ float d_mask[MASK_DIM_SIZE * MASK_DIM_SIZE];


/*-------------------------------------------------------------------------------------------------
 * FORWARD DECLARATION
 *-----------------------------------------------------------------------------------------------*/
__global__ void naive_convolution_kernel(const float *d_in, float *d_out,
        int M, int N, int mask_m, int mask_n);
__global__ void tiled_convolution_kernel(const float *d_in, float *d_out,
        int M, int N, int mask_m, int mask_n);
__global__ void cache_convolution_kernel(const float *d_in, float *d_out,
        int M, int N, int mask_m, int mask_n);

void cpu_convolution(const float *A, const float *mask, float *out,
        int M, int N, int mask_m, int mask_n);
float convolution_naive(const float *in, float *out, int M, int N, int mask_m, int mask_n);
float convolution_tiled(const float *in, float *out, int M, int N, int mask_m, int mask_n);
float convolution_cache(const float *in, float *out, int M, int N, int mask_m, int mask_n);
float check_convolution(const float *A, const float *B, int M, int N);


/*-------------------------------------------------------------------------------------------------
 * MAIN
 *-----------------------------------------------------------------------------------------------*/
int main(int argc, char **argv)
{
    const int M = 2000, N = 2000;
    const int mask_m = 3, mask_n = 3;
    float *A = new float[M * N];
    float *cpu_out = new float[M * N];
    float *naive_out = new float[M * N];
    float *tiled_out = new float[M * N];
    float *cache_out = new float[M * N];
    float *mask = new float[mask_m * mask_n];

    std::random_device rd;
    std::mt19937 engine(rd());
    std::uniform_real_distribution<float> rand0(0, 10.0);

    for (int i = 0; i < M * N; i++)
        A[i] = rand0(engine);

    mask[0] = 1.0;
    mask[1] = 2.0;
    mask[2] = 1.0;

    mask[3] = 2.0;
    mask[4] = 4.0;
    mask[5] = 2.0;

    mask[6] = 1.0;
    mask[7] = 2.0;
    mask[8] = 1.0;

    hipMemcpyToSymbol(HIP_SYMBOL(d_mask), mask, mask_m * mask_n * sizeof(float));

    cpu_convolution(A, mask, cpu_out, M, N, mask_m, mask_n);
    float naive_time = convolution_naive(A, naive_out, M, N, mask_m, mask_n);
    float tiled_time = convolution_tiled(A, tiled_out, M, N, mask_m, mask_n);
    float cache_time = convolution_cache(A, cache_out, M, N, mask_m, mask_n);

    float naive_err = check_convolution(cpu_out, naive_out, M, N);
    std::cout << "Naive convolution error = " << naive_err << ". Time = " << naive_time << "ms.\n";

    float tiled_err = check_convolution(cpu_out, tiled_out, M, N);
    std::cout << "tiled convolution error = " << tiled_err<< ". Time = " << tiled_time << "ms.\n";

    float cache_err = check_convolution(cpu_out, cache_out, M, N);
    std::cout << "cache convolution error = " << cache_err << ". Time = " << cache_time << "ms.\n";

    delete[] A;
    delete[] cpu_out;
    delete[] naive_out;
    delete[] tiled_out;
    delete[] cache_out;
    delete[] mask;
}


/*-------------------------------------------------------------------------------------------------
 * KERNELS
 *-----------------------------------------------------------------------------------------------*/

__global__ void naive_convolution_kernel(const float *d_in, float *d_out,
        int M, int N, int mask_m, int mask_n)
{
    int m_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int n_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if indices are within matrix
    if (m_idx < M && n_idx < N) {
        int start_m = m_idx - (mask_m / 2);
        int start_n = n_idx - (mask_n / 2);
        float val = 0.0;

        // Loop over mask
        for (int m = 0; m < mask_m; m++) {
            for (int n = 0; n < mask_n; n++) {
                int i = start_m + m;
                int j = start_n + n;

                if (i >= 0 && i < M && j >= 0 && j < N)
                    val += d_in[i * N + j] * d_mask[m * mask_n + n];
            }
        }

        d_out[m_idx * N + n_idx] = val;
    }
}


__global__ void tiled_convolution_kernel(const float *d_in, float *d_out,
        int M, int N, int mask_m, int mask_n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int m_half = mask_m / 2;
    int n_half = mask_n / 2;

    __shared__ float tile[TILE_SIZE + MASK_DIM_SIZE - 1][TILE_SIZE + MASK_DIM_SIZE - 1];

    int left_idx = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    int right_idx = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
    int top_idx = (blockIdx.y - 1) * blockDim.y + threadIdx.y;
    int bottom_idx = (blockIdx.y + 1) * blockDim.y + threadIdx.y;

    // Load block matrix
    tile[threadIdx.y + m_half][threadIdx.x + n_half] = (row < M || col < N) ? d_in[row * N + col] : 0;

    // Load left and right tiles
    if (threadIdx.x >= blockDim.x - n_half)
        tile[threadIdx.y + m_half][threadIdx.x - blockDim.x + n_half] =
            (left_idx < 0) ? 0 : d_in[row * N + left_idx];

    if (threadIdx.x < n_half)
        tile[threadIdx.y + m_half][threadIdx.x + blockDim.x + n_half] =
            (right_idx >= N) ? 0 : d_in[row * N + right_idx];

    // Load top and bottom tiles, including corners
    if (threadIdx.y >= blockDim.y - m_half) {
        tile[threadIdx.y - blockDim.y + m_half][threadIdx.x + n_half] =
            (top_idx < 0) ? 0 : d_in[top_idx * N + col];

        if (threadIdx.x >= blockDim.x - n_half)
            tile[threadIdx.y - blockDim.y + m_half][threadIdx.x - blockDim.x + n_half] =
                (top_idx < 0 || left_idx < 0) ? 0 : d_in[top_idx * N + left_idx];
        else if (threadIdx.x < n_half)
            tile[threadIdx.y - blockDim.y + m_half][threadIdx.x + blockDim.x + n_half] =
                (top_idx < 0 || right_idx >= N) ? 0 : d_in[top_idx * N + right_idx];
    }

    if (threadIdx.y < m_half) {
        tile[threadIdx.y + blockDim.y + m_half][threadIdx.x + n_half] =
            (bottom_idx >= M) ? 0 : d_in[bottom_idx * N + col];

        if (threadIdx.x >= blockDim.x - n_half)
            tile[threadIdx.y + blockDim.y + m_half][threadIdx.x - blockDim.x + n_half] =
                (bottom_idx >= M || left_idx < 0) ? 0 : d_in[bottom_idx * N + left_idx];
        else if (threadIdx.x < n_half)
            tile[threadIdx.y + blockDim.y + m_half][threadIdx.x + blockDim.x + n_half] =
                (bottom_idx >= M || right_idx >= N) ? 0 : d_in[bottom_idx * N + right_idx];
    }

    __syncthreads();

    if (row < M && col < N) {
        float sum = 0.0;

        // Perform convolution
        for (int i = 0; i < mask_m; i++)
            for (int j = 0; j < mask_n; j++)
                sum += tile[threadIdx.y + i][threadIdx.x + j] * d_mask[i * mask_n + j];

        d_out[row * N + col] = sum;
    }
}


__global__ void cache_convolution_kernel(const float *d_in, float *d_out,
        int M, int N, int mask_m, int mask_n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int m_half = mask_m / 2;
    int n_half = mask_n / 2;

    __shared__ float tile[TILE_SIZE][TILE_SIZE];

    if (row < M && col < N)
        tile[threadIdx.y][threadIdx.x] = d_in[row * N + col];

    __syncthreads();

    int row_tile_start = blockIdx.y * blockDim.y;
    int row_tile_end = (blockIdx.y + 1) * blockDim.y;
    int col_tile_start = blockIdx.x * blockDim.x;
    int col_tile_end = (blockIdx.x + 1) * blockDim.x;

    int m_start = row - m_half;
    int n_start = col - n_half;
    float sum = 0.0;

    if (row < M && col < N) {
        // Loop over mask
        for (int i = 0; i < mask_m; i++) {
            for (int j = 0; j < mask_n; j++) {
                int m = m_start + i;
                int n = n_start + j;

                // Check if indicies with mask applied is within matrix
                if (m >= 0 && m < M && n >= 0 && n < N) {
                    // Check if elements are in the cache
                    if (m >= row_tile_start && m < row_tile_end &&
                            n >= col_tile_start && n < col_tile_end)
                        sum += tile[threadIdx.y + i - m_half][threadIdx.x + j - n_half] *
                            d_mask[i * mask_n + j];
                    else
                        sum += d_in[m * N + n] * d_mask[i * mask_n + j];
                }
            }
        }
        d_out[row * N + col] = sum;
    }
}


/*-------------------------------------------------------------------------------------------------
 * FUNCTIONS
 *-----------------------------------------------------------------------------------------------*/

void cpu_convolution(const float *A, const float *mask, float *out,
        int M, int N, int mask_m, int mask_n)
{
    int half_m = mask_m / 2;
    int half_n = mask_n / 2;

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {

            int start_m = i - half_m;
            int start_n = j - half_n;
            float val = 0.0;

            for (int m = 0; m < mask_m; m++) {
                for (int n = 0; n < mask_n; n++) {
                    int m_idx = start_m + m;
                    int n_idx = start_n + n;

                    if (m_idx >= 0 && m_idx < M && n_idx >= 0 && n_idx < N)
                        val += A[m_idx * N + n_idx] * mask[m * mask_n + n];
                }
            }
            out[i * N + j] = val;
        }
    }
}


float convolution_naive(const float *in, float *out, int M, int N, int mask_m, int mask_n)
{
    int size = M * N * sizeof(float);
    float *d_in, *d_out;

    hipMalloc((void**) &d_in, size);
    hipMalloc((void**) &d_out, size);

    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    int N_thd = TILE_SIZE;
    int M_blk = ceil(static_cast<float>(M) / static_cast<float>(N_thd));
    int N_blk = ceil(static_cast<float>(N) / static_cast<float>(N_thd));

    dim3 grid_dim(M_blk, N_blk, 1);
    dim3 blk_dim(N_thd, N_thd, 1);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    naive_convolution_kernel<<<grid_dim, blk_dim>>>(d_in, d_out, M, N, mask_m, mask_n);
    hipEventRecord(stop);

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);

    hipFree(d_in);
    hipFree(d_out);

    return time_ms;
}


float convolution_tiled(const float *in, float *out, int M, int N, int mask_m, int mask_n)
{
    int size = M * N * sizeof(float);
    float *d_in, *d_out;

    hipMalloc((void**) &d_in, size);
    hipMalloc((void**) &d_out, size);

    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    int N_thd = TILE_SIZE;
    int M_blk = ceil(static_cast<float>(M) / static_cast<float>(N_thd));
    int N_blk = ceil(static_cast<float>(N) / static_cast<float>(N_thd));

    dim3 grid_dim(M_blk, N_blk, 1);
    dim3 blk_dim(N_thd, N_thd, 1);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    tiled_convolution_kernel<<<grid_dim, blk_dim>>>(d_in, d_out, M, N, mask_m, mask_n);
    hipEventRecord(stop);

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);

    hipFree(d_in);
    hipFree(d_out);

    return time_ms;
}


float convolution_cache(const float *in, float *out, int M, int N, int mask_m, int mask_n)
{
    int size = M * N * sizeof(float);
    float *d_in, *d_out;

    hipMalloc((void**) &d_in, size);
    hipMalloc((void**) &d_out, size);

    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    int N_thd = TILE_SIZE;
    int M_blk = ceil(static_cast<float>(M) / static_cast<float>(N_thd));
    int N_blk = ceil(static_cast<float>(N) / static_cast<float>(N_thd));

    dim3 grid_dim(M_blk, N_blk, 1);
    dim3 blk_dim(N_thd, N_thd, 1);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    cache_convolution_kernel<<<grid_dim, blk_dim>>>(d_in, d_out, M, N, mask_m, mask_n);
    hipEventRecord(stop);

    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);

    hipFree(d_in);
    hipFree(d_out);

    return time_ms;
}


float check_convolution(const float *A, const float *B, int M, int N)
{
    float sum = 0.0;

    for (int i = 0; i < N; i++)
        sum += (A[i] - B[i]) * (A[i] - B[i]);

    return sqrt(sum);
}
