#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include <openblas/cblas.h>

#include "sgemm_kernel.h"


float compute_error(const float *A, const float *B, int N);


int main(void)
{
    const int N = 4096;
    float *A     = new float[N * N];
    float *B     = new float[N * N];
    float *C_gpu = new float[N * N];
    float *C_ref = new float[N * N];

    for (int i = 0; i < N * N; i++) {
        A[i] = static_cast<float>(i % 50);
        B[i] = static_cast<float>(i % 10 + 1.0);
        C_ref[i] = 0.0;
    }

    // Compute reference on the CPU
    auto start = std::chrono::high_resolution_clock::now();
    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, N, N, N, 1.0, A, N, B, N, 0.0, C_ref, N);
    auto end = std::chrono::high_resolution_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "Reference (openBLAS) runtime: " << elapsed << "ms.\n";

    sgemm_basic(A, B, C_gpu, N);
    std::cout << " Error per element = " << compute_error(C_gpu, C_ref, N) << '\n';

    sgemm_tiled(A, B, C_gpu, N);
    std::cout << " Error per element = " << compute_error(C_gpu, C_ref, N) << '\n';

    delete[] A;
    delete[] B;
    delete[] C_gpu;
    delete[] C_ref;
}


float compute_error(const float *A, const float *B, int N)
{
    float error = 0.0;

    for (int i = 0; i < N * N; i++)
        error += fabs(A[i] - B[i]);

    return error / static_cast<float>(N * N);
}
