#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>

#include "sgemm_kernel.h"


/*-------------------------------------------------------------------------------------------------
 * KERNELS
 *-----------------------------------------------------------------------------------------------*/

/* sgemm_basic_kernel()
 * Basic SGEMM kernel where each thread performs a corresponding dot product.
 */
__global__ void sgemm_basic_kernel(const float *A, const float *B, float *C, int N)
{
    // Compute the row and col indices
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if ((r < N) && (c < N)) {
        float sum = 0.0;

        for (int i = 0; i < N; i++)
            sum += A[r * N + i] * B[i * N + c];

        C[r * N + c] = sum;
    } // Check if row and col indices is a part of the matrix
}


/* sgemm_tiled_kernel()
 * SGEMM with tiling. Sepereates matrix into  TILE_WIDTH x TILE_WIDTH submatrices and performs a
 * dot product on the row and column in the submatrix.
 */
__global__ void sgemm_tiled_kernel(const float *A, const float *B, float *C, int N)
{
    __shared__ float A_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_tile[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x,  by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    // Compute row and column indices
    int r  = by * TILE_WIDTH + ty;
    int c  = bx * TILE_WIDTH + tx;
    float sum = 0.0;

    for (int i = 0; i < N / TILE_WIDTH; i++) {
        A_tile[ty][tx] = A[r * N + i * TILE_WIDTH + tx];
        B_tile[ty][tx] = B[(i * TILE_WIDTH + ty) * N + c];
        __syncthreads();

        // Perform dot product for C[r][c] using the block
        #pragma unroll
        for (int k = 0; k < TILE_WIDTH; k++)
            sum += A_tile[ty][k] * B_tile[k][tx];
        __syncthreads();
    } // Loop over all tiles neede for matrix multiply

    C[r * N + c] = sum;
}


/*-------------------------------------------------------------------------------------------------
 * FUNCTIONS
 *-----------------------------------------------------------------------------------------------*/

/* sgemm_basic()
 * Function for calling SGEMM.
 */
void sgemm_basic(const float *A, const float *B, float *C, int N)
{
    int size = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int N_thd = 16;
    int N_blk = ceil(static_cast<float>(N) / static_cast<float>(N_thd));
    dim3 grid_dim(N_blk, N_blk, 1);
    dim3 blk_dim(N_thd, N_thd, 1);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sgemm_basic_kernel<<<grid_dim, blk_dim>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float time_ms = 0.0;
    hipEventElapsedTime(&time_ms, start, stop);

    std::cout << "Basic GPU SGEMM runtime: " << time_ms << "ms ";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


/* sgemm_tiled()
 * Function for calling SGEMM.
 */
void sgemm_tiled(const float *A, const float *B, float *C, int N)
{
    int size = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int N_thd = 32;
    int N_blk = ceil(static_cast<float>(N) / static_cast<float>(N_thd));
    dim3 grid_dim(N_blk, N_blk, 1);
    dim3 blk_dim(N_thd, N_thd, 1);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sgemm_tiled_kernel<<<grid_dim, blk_dim>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float time_ms = 0.0;
    hipEventElapsedTime(&time_ms, start, stop);

    std::cout << "Tiled GPU SGEMM runtime: " << time_ms << "ms ";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
