#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>

#include "sgemm_kernel.h"


/*-------------------------------------------------------------------------------------------------
 * KERNELS
 *-----------------------------------------------------------------------------------------------*/

/* sgemm_basic_kernel()
 * Basic SGEMM kernel where each thread performs a corresponding dot product.
 */
__global__ void sgemm_basic_kernel(const float *A, const float *B, float *C, int N)
{
    // Compute the row and col indices
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if ((r < N) && (c < N)) {
        float sum = 0.0;

        for (int i = 0; i < N; i++)
            sum += A[r * N + i] * B[i * N + c];

        C[r * N + c] = sum;
    } // Check if row and col indices is a part of the matrix
}


/*-------------------------------------------------------------------------------------------------
 * FUNCTIONS
 *-----------------------------------------------------------------------------------------------*/

/* sgemm_basic()
 * Function for calling SGEMM.
 */
void sgemm_basic(const float *A, const float *B, float *C, int N)
{
    int size = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    int N_thd = 16;
    int N_blk = ceil(static_cast<float>(N) / static_cast<float>(N_thd));
    dim3 grid_dim(N_blk, N_blk, 1);
    dim3 blk_dim(N_thd, N_thd, 1);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sgemm_basic_kernel<<<grid_dim, blk_dim>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float time_ms = 0.0;
    hipEventElapsedTime(&time_ms, start, stop);

    std::cout << "Basic GPU SGEMM runtime: " << time_ms << "ms ";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
