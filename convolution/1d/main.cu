/* Written by : Eric Tan
 */

#include <iostream>
#include <cmath>
#include <array>
#include <hip/hip_runtime.h>

#define MAX_MASK_SIZE 50
#define TILE_SIZE 512
#define N_TILE 4


/*-------------------------------------------------------------------------------------------------
 * GLOBAL CONSTANTS
 *-----------------------------------------------------------------------------------------------*/
__constant__ float mask[MAX_MASK_SIZE];


/*-------------------------------------------------------------------------------------------------
 * FORWARD DECLARATION
 *-----------------------------------------------------------------------------------------------*/
__global__ void tiled_convolution_kernel(const float *input, float *output, int N_data, int N_mask);
__global__ void cache_convolution_kernel(const float *input, float *output, int N_data, int N_mask);
void convolution_tiled_gpu(const float *input, float *output, int N_data, int N_mask);
void convolution_cache_gpu(const float *input, float *output, int N_data, int N_mask);
void convolution_cpu(const float *input, float *output, const float *mask, int N_data, int N_mask);
double check_convolution(const float *A, const float *B, int N);


/*-------------------------------------------------------------------------------------------------
 * MAIN
 *-----------------------------------------------------------------------------------------------*/
int main(void)
{
    const int mask_size = 9;
    const int N_data = 1000;
    const std::array<float, mask_size> mask_kernel = {3.0, 4.0, 5.0, 6.0, 7.0, 6.0, 5.0, 4.0, 3.0};

    // Copy to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(mask), mask_kernel.data(), mask_size * sizeof(float));

    float *input = new float[N_data];
    float *cpu_output = new float[N_data];
    float *gpu_output = new float[N_data];

    for (int i = 0; i < N_data; i++)
        input[i] = i + 1.0;

    convolution_cpu(input, cpu_output, mask_kernel.data(), N_data, mask_size);

    convolution_tiled_gpu(input, gpu_output, N_data, mask_size);
    std::cout << " Error = " << check_convolution(cpu_output, gpu_output, N_data) << '\n';

    convolution_cache_gpu(input, gpu_output, N_data, mask_size);
    std::cout << " Error = " << check_convolution(cpu_output, gpu_output, N_data) << '\n';

    delete[] input;
    delete[] cpu_output;
    delete[] gpu_output;
}


/*-------------------------------------------------------------------------------------------------
 * KERNELS
 *-----------------------------------------------------------------------------------------------*/

/* tiled_convolution_kernel()
 * Performs convolution by tiling the input and loading halo elements.
 */
__global__ void tiled_convolution_kernel(const float *input, float *output, int N_data, int N_mask)
{
    int idx  = blockIdx.x * blockDim.x + threadIdx.x;
    int half = N_mask / 2;
    __shared__ float tile[TILE_SIZE + MAX_MASK_SIZE - 1];

    // Left load
    // Maps the last N_mask/2 threads to the previous block (or 0 if at the edge)
    int left_idx = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x >= blockDim.x - half)
        tile[threadIdx.x + half - blockDim.x] = (left_idx < 0) ? 0 : input[left_idx];

    // Middle load
    tile[half + threadIdx.x] = input[blockIdx.x * blockDim.x + threadIdx.x];

    // Right load
    // Maps the first N_mask/2 threads to the next block (or 0 if at the edge)
    int right_idx = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x < half)
        tile[threadIdx.x + blockDim.x + half] = (right_idx >= N_data) ? 0 : input[right_idx];

    __syncthreads();

    // Convolution calculation
    float sum = 0.0;
    for (int i = 0; i < N_mask; i++)
        sum += tile[threadIdx.x + i] * mask[i];

    output[idx] = sum;
}


/* cache_convolution_kernel()
 * Performs convolution by loading multiple tiles of the input data and performing the convolution
 * operation. Halo elements are assumed to be loaded in the L2 cache so if theya re needed, we
 * will go to L2 cache to grabe them.
 */
__global__ void cache_convolution_kernel(const float *input, float *output, int N_data, int N_mask)
{
    int idx = (N_TILE * blockDim.x * blockIdx.x) + threadIdx.x;
    __shared__ float tile[N_TILE * TILE_SIZE];

    // Load Multiple tiles
    #pragma unroll
    for (int i = 0; i < N_TILE; i++)
        tile[threadIdx.x + i * TILE_SIZE] = input[idx + i * TILE_SIZE];

    __syncthreads();

    int tile_start = N_TILE * blockDim.x * blockIdx.x;
    int tile_end   = N_TILE * blockDim.x * (blockIdx.x + 1);

    #pragma unroll
    for (int i = 0; i < N_TILE; i++) {
        int start_point = idx + (i * TILE_SIZE) - (N_mask / 2);
        float sum = 0.0;

        for (int j = 0; j < N_mask; j++) {
            int input_idx = start_point + j;

            if (input_idx >= 0 && input_idx < N_data) {
                // Check if elements needed is in shared memory
                if (input_idx >= tile_start && input_idx < tile_end)
                    sum += tile[threadIdx.x + j - (N_mask / 2) + (i * TILE_SIZE)] * mask[j];
                else
                    sum += input[input_idx] * mask[j];

            } // Check if element is outside range of input data
        } // Loop over Mask elements

        output[idx + i * TILE_SIZE] = sum;
    } // Loop over multiple tiles
}


/*-------------------------------------------------------------------------------------------------
 * FUNCTIONS
 *-----------------------------------------------------------------------------------------------*/

/* convolution_tiled_gpu()
 * Driver function for tiled convolution which loads halo elements.
 */
//void convolution_tiled_gpu(const float *input, float *output, int N_data, int N_mask)
void convolution_tiled_gpu(const float *input, float *output, int N_data, int N_mask)
{
    int size = N_data * sizeof(float);
    float *d_input, *d_output;

    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    int N_thd = TILE_SIZE;
    int N_blk = ceil(static_cast<float>(N_data) / static_cast<float>(N_thd));
    dim3 blk_dim(N_thd, 1, 1);
    dim3 grid_dim(N_blk, 1, 1 );

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    tiled_convolution_kernel<<<grid_dim, blk_dim>>>(d_input, d_output, N_data, N_mask);

    hipEventRecord(stop);
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float time_ms = 0.0;
    hipEventElapsedTime(&time_ms, start, stop);

    std::cout << "Convolution with tiling and halo runtime: " << time_ms << "ms ";

    hipFree(d_input);
    hipFree(d_output);
}


/* convolution_tiled_gpu()
 * Driver function for tiled convolution without loading halo elements.
 */
void convolution_cache_gpu(const float *input, float *output, int N_data, int N_mask)
{
    int size = N_data * sizeof(float);
    float *d_input, *d_output;

    hipMalloc((void**)&d_input, size);
    hipMalloc((void**)&d_output, size);

    hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);

    int N_thd = TILE_SIZE;
    int N_blk = ceil(static_cast<float>(N_data) / static_cast<float>(N_TILE * N_thd));
    dim3 blk_dim(N_thd, 1, 1);
    dim3 grid_dim(N_blk, 1, 1 );

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    cache_convolution_kernel<<<grid_dim, blk_dim>>>(d_input, d_output, N_data, N_mask);

    hipEventRecord(stop);
    hipMemcpy(output, d_output, size, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float time_ms = 0.0;
    hipEventElapsedTime(&time_ms, start, stop);

    std::cout << "Convolution with L2 cache use runtime: " << time_ms << "ms ";

    hipFree(d_input);
    hipFree(d_output);
}


/* convolution_cpu()
 * CPU based convolution operation. Used to compare GPU implamentations.
 */
void convolution_cpu(const float *input, float *output, const float *mask, int N_data, int N_mask)
{
    int half_point = N_mask / 2;

    for (int i = 0; i < N_data; i++) {
        float sum = 0.0;
        for (int j = 0; j < N_mask; j++) {
            if (i - half_point + j >= 0 && i - half_point + j < N_data)
                sum += input[i - half_point + j] * mask[j];
        }
        output[i] = sum;
    }
}


/* check_convolution()
 * Computes the difference between the two outputs of CPU and GPU convolutions.
 */
double check_convolution(const float *A, const float *B, int N)
{
    float sum = 0.0;

    for (int i = 0; i < N; i++)
        sum += fabs(A[i] - B[i]);

    return sum;
}
